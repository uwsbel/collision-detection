#include "hip/hip_runtime.h"
#include "Broadphase.cuh"
#include <assert.h>

// Test to see how many bins can be created. There will only be two objects in each bin, so the number of potential collisions will equal the number of bins.

struct Sphere {
    double r;
    real3 pos;
    uint index;
};

int main(int argc, char** argv)
{
	// Step 1: Generate random spheres
	uint numSpheresPerSide = 10; // number of spheres per side
	if(argc==2) numSpheresPerSide = atoi(argv[1]);
	double sphereRad = 0.25;
	double sphereSpacing = 1;

	vector<Sphere> spheres;

	Sphere sphere;
	int index = 0;
	for(int k=0;k<numSpheresPerSide;k++)
	{
		for(int j=0;j<numSpheresPerSide;j++)
		{
			for(int i=0;i<numSpheresPerSide;i++)
			{
				sphere.pos = make_real3(i*sphereSpacing,j*sphereSpacing,k*sphereSpacing);
				sphere.r = sphereSpacing;
				sphere.index = index;

				// add two of these spheres so they must collide with eachother
				spheres.push_back(sphere);
				index++;
				spheres.push_back(sphere);
				index++;
			}
		}
	}
	// End Step 1

	// Step 2: Generate aabb_data from spheres
	custom_vector<real3> aabb_data;

	for(int i=0;i<spheres.size();i++)
	{
		real3 temp = spheres[i].pos;
		temp.x-=spheres[i].r;
		temp.y-=spheres[i].r;
		temp.z-=spheres[i].r;
		aabb_data.push_back(temp);
	}
	for(int i=0;i<spheres.size();i++)
	{
		real3 temp = spheres[i].pos;
		temp.x+=spheres[i].r;
		temp.y+=spheres[i].r;
		temp.z+=spheres[i].r;
		aabb_data.push_back(temp);
	}
	// End Step 2

	// Step 3: Run broadphase algorithm to find potential collisions
	custom_vector<long long> potentialCollisions;

	Broadphase broadphaseManager;
	broadphaseManager.setBinsPerAxis(make_real3(numSpheresPerSide,numSpheresPerSide,numSpheresPerSide));

	cout << "Begin parallel broadphase" << endl;
	double startTime = omp_get_wtime();
	broadphaseManager.detectPossibleCollisions(aabb_data, potentialCollisions);
	double endTime = omp_get_wtime();
	printf("Time to detect: %lf seconds (%d possible collisions)\n", (endTime - startTime),broadphaseManager.getNumPossibleContacts());
	cout << "End parallel broadphase\n" << endl;
	// End Step 3

	assert(broadphaseManager.getNumPossibleContacts()==pow(numSpheresPerSide,3));
	cout << "TEST PASSED" << endl;

	return 0;
}
